#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void print_idx_kernel(){
    printf("block idx: (%3d, %3d, %3d), thread idx: (%3d, %3d, %3d)\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         threadIdx.z, threadIdx.y, threadIdx.x);
}

__global__ void print_dim_kernel(){
    printf("grid dimension: (%3d, %3d, %3d), block dimension: (%3d, %3d, %3d)\n",
         gridDim.z, gridDim.y, gridDim.x,
         blockDim.z, blockDim.y, blockDim.x);
}

// 在block空间中打印每个thread一维连续的线性索引
__global__ void print_thread_idx_per_block_kernel(){
    int index = threadIdx.z * blockDim.x * blockDim.y + \
              threadIdx.y * blockDim.x + \
              threadIdx.x;

    printf("block idx: (%3d, %3d, %3d), thread idx: %3d\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         index);
}

// 在grid空间中打印每个thread一维连续的线性索引
__global__ void print_thread_idx_per_grid_kernel(){
    int bSize  = blockDim.z * blockDim.y * blockDim.x;

    int bIndex = blockIdx.z * gridDim.x * gridDim.y + \
               blockIdx.y * gridDim.x + \
               blockIdx.x;

    int tIndex = threadIdx.z * blockDim.x * blockDim.y + \
               threadIdx.y * blockDim.x + \
               threadIdx.x;

    int index  = bIndex * bSize + tIndex;//从这里开始理解

    printf("block idx: %3d, thread idx in block: %3d, thread idx: %3d\n", 
         bIndex, tIndex, index);
}


__global__ void print_cord_kernel(){
    int index = threadIdx.z * blockDim.x * blockDim.y + \
              threadIdx.y * blockDim.x + \
              threadIdx.x;

    int x  = blockIdx.x * blockDim.x + threadIdx.x;
    int y  = blockIdx.y * blockDim.y + threadIdx.y;

    printf("block idx: (%3d, %3d, %3d), thread idx: %3d, cord: (%3d, %3d)\n",
         blockIdx.z, blockIdx.y, blockIdx.x,
         index, x, y);
}

void print_one_dim(){
    int inputSize = 8;//1*8大小的向量
    int blockDim = 4;
    int gridDim = inputSize / blockDim;

    dim3 block(blockDim);
    dim3 grid(gridDim);

    // print_idx_kernel<<<grid, block>>>();
    // print_dim_kernel<<<grid, block>>>();
    // print_thread_idx_per_block_kernel<<<grid, block>>>();
    print_thread_idx_per_grid_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

void print_two_dim(){
    int inputWidth = 4;//4*4大小的矩阵
    int blockDim = 2;
    int gridDim = inputWidth / blockDim;

    dim3 block(blockDim, blockDim);
    dim3 grid(gridDim, gridDim);

    print_idx_kernel<<<grid, block>>>();
    print_dim_kernel<<<grid, block>>>();
    // print_thread_idx_per_block_kernel<<<grid, block>>>();
    print_thread_idx_per_grid_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

void print_three_dim(){
    int inputWidth = 4;//4*4*4大小的立方体
    int blockDim = 2;
    int gridDim = inputWidth / blockDim;

    dim3 block(blockDim, blockDim, blockDim);
    dim3 grid(gridDim, gridDim, gridDim);

    print_idx_kernel<<<grid, block>>>();
    print_dim_kernel<<<grid, block>>>();
    // print_thread_idx_per_block_kernel<<<grid, block>>>();
    print_thread_idx_per_grid_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

void print_cord(){
    int inputWidth = 4;//一张4*4大小的图片

    int blockDim = 2;
    int gridDim = inputWidth / blockDim;

    dim3 block(blockDim, blockDim);
    dim3 grid(gridDim, gridDim);

    print_cord_kernel<<<grid, block>>>();

    hipDeviceSynchronize();
}

int main() {
    /*
    synchronize是同步的意思，有几种synchronize

    cudaDeviceSynchronize: CPU与GPU端完成同步，CPU不执行之后的语句，知道这个语句以前的所有cuda操作结束
    cudaStreamSynchronize: 跟cudaDeviceSynchronize很像，但是这个是针对某一个stream的。只同步指定的stream中的cpu/gpu操作，其他的不管
    cudaThreadSynchronize: 现在已经不被推荐使用的方法
    __syncthreads:         线程块内同步
    */
    // print_one_dim();
    print_two_dim();
    // print_three_dim();
    print_cord();
    return 0;
}
